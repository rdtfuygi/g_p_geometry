#include "hip/hip_runtime.h"
#include "geometry.cuh"
#define _USE_MATH_DEFINES 
#include <math.h>


tirangle::tirangle()
{
	segs[0].origin = point(0.0f, 0.0f);
	segs[1].origin = point(1.0f, 0.0f);
	segs[2].origin = point(0.0f, 1.0f);
	segs[0].dir = vector(1.0f, 0.0f);
	segs[0].dir = vector(-float(M_SQRT1_2), float(M_SQRT1_2));
	segs[0].dir = vector(0.0f, -1.0f);
}

tirangle::tirangle(const point* ��)
{
	segs[0] = seg(��[0], ��[1]);
	segs[1] = seg(��[1], ��[2]);
	segs[2] = seg(��[2], ��[0]);
}

tirangle::tirangle(std::vector<point>& ��)
{
	segs[0] = seg(��[0], ��[1]);
	segs[1] = seg(��[1], ��[2]);
	segs[2] = seg(��[2], ��[0]);
}

__host__ __device__ seg& tirangle::operator[](int i)
{
	return segs[i];
}

__host__ __device__ seg tirangle::operator[](int i) const
{
	return segs[i];
}


__host__ __device__ void tirangle::reset_seg()
{
	for (int i = 0; i < 3; i++)
	{
		reset_seg(i);
	}
}

__host__ __device__ void tirangle::reset_seg(int i)
{
	segs[i] = seg(segs[i].origin, segs[(i + 1) % 3].origin);
}

__host__ __device__ bool tirangle::is_cross(const seg l) const
{
	return (::is_cross(segs[0], l)) || (::is_cross(segs[1], l)) || (::is_cross(segs[2], l));
}

__host__ __device__ float tirangle::area() const
{
	return abs((vector(segs[1].origin) - vector(segs[0].origin)) ^ (vector(segs[2].origin) - vector(segs[0].origin))) / 2;
}
#ifndef no_opencv
void tirangle::print(cv::InputOutputArray ͼ��, float ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	seg(segs[0].origin, segs[1].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	seg(segs[1].origin, segs[2].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	seg(segs[2].origin, segs[0].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
}
#endif