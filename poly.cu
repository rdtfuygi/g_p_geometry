#include "hip/hip_runtime.h"
#include "geometry.cuh"



__host__ __device__ void sort(float* list, int n, bool up = true)
{
	for (int i = n - 1; i > 0; i--)
	{
		bool swap = false;
		for (int j = 0; j < i; j++)
		{
			if ((list[j] > list[j + 1]) && up)
			{
				float temp_dist = list[j];
				list[j] = list[j + 1];
				list[j + 1] = temp_dist;
				swap = true;
			}
			else if ((list[j] < list[j + 1]) && !up)
			{
				float temp_dist = list[j];
				list[j] = list[j + 1];
				list[j + 1] = temp_dist;
				swap = true;
			}
		}
		if (!swap)
		{
			break;
		}
	}
}



__host__ __device__ void poly::changed() const
{
	area_change = true;
	dir_area_change = true;
	center_change = true;
	fast_center_change = true;
	legal_change = true;
	seg_change = true;
}

__host__ __device__ poly::poly()
{
	changed();
}

__host__ __device__ poly::poly(const point* ��, int m)
{
	int temp = m < 20 ? m : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp; i < 20 ; i++)
	{
		segs[i] = seg(��[temp], ��[temp + 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);

	changed();
}

poly::poly(std::vector<point>& ��)
{
	int temp = (��.size() < 20 ) ? ��.size() : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp - 1; i < 20 - 1; i++)
	{
		segs[i] = seg(��[��.size() - 1], ��[��.size() - 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);

	changed();
}

poly::poly(const tirangle ����)
{
	segs[0] = ����.segs[0];
	segs[1] = ����.segs[1];
	segs[2] = ����.segs[2];
	reset_seg();

	changed();
}

__host__ __device__ bool poly::legal() const
{
	if (!legal_change)
	{
		return legal_;
	}
	legal_change = false;

	for (int i = 1; i < 19; i++)
	{
		for (int j = 0; j < i - 1; j++)
		{
			if ((segs[i].dist < 0.001f) || (segs[j].dist < 0.001f))
			{
				continue;
			}
			float t1, t2;
			cross(segs[i], segs[j], t1, t2);
			if ((t2 > 0.001f) && (t2 < segs[j].dist - 0.001f))
			{
				legal_= false;
				return false;
			}
		}
	}
	for (int j = 1; j < 18; j++)
	{
		if ((segs[19].dist < 0.001f) || (segs[j].dist < 0.001f))
		{
			continue;
		}
		float t1, t2;
		cross(segs[19], segs[j], t1, t2);
		if ((t2 > 0.001) && (t2 < segs[j].dist - 0.001f))
		{
			legal_ = false;
			return false;
		}
	}
	legal_ = true;
	return true;
}

//__host__ __device__ float poly::one_link_area()
//{
//	point max = segs[0].origin, min = segs[0].origin;
//	for (int i = 1; i < 20; i++)
//	{
//		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
//		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
//		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
//		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
//	}
//
//
//	float last[20];
//	{
//		ray temp;
//		temp.origin = point(int(min[0] + 1), min[1]);
//		temp.dir = vector(0.0, 1.0);
//		for (int i = 0; i < 20; i++)
//		{
//			float t_1, t_2;
//			cross(temp, segs[i], t_1, t_2);
//			last[i] = t_1;
//		}
//		for (int i = 19; i > 0; i--)
//		{
//			bool swap = false;
//			for (int j = 0; j < i; j++)
//			{
//				if (last[j] < last[j + 1])
//				{
//					continue;
//				}
//				float temp_dist = last[j];
//				last[j] = last[j + 1];
//				last[j + 1] = temp_dist;
//				swap = true;
//			}
//			if (!swap)
//			{
//				break;
//			}
//		}
//	}
//
//
//	float areas[10];
//	char map[10] = { 0,1,2,3,4,5,6,7,8,9 };
//	for (int i = 0; i < 10; i++)
//	{
//		if ((last[2 * i + 1] != FLT_MAX) && (last[2 * i] != FLT_MAX))
//		{
//			areas[i] = last[2 * i + 1] - last[2 * i];
//		}
//		else
//		{
//			areas[i] = 0;
//		}
//	}
//
//	for (int x = min[0] + 2; x < max[0]; x++)
//	{
//		float dist[20];
//		char map_new[10] = { 10,10,10,10,10,10,10,10,10,10 };
//
//		seg temp;
//		temp.origin = point(x, min[1]);
//		temp.dir = vector(0.0, 1.0);
//		temp.dist = max[1] - min[1];
//		for (int i = 0; i < 20; i++)
//		{
//			float t_1, t_2;
//			cross(temp, segs[i], t_1, t_2);
//			dist[i] = t_1;
//		}
//		for (int i = 19; i > 0; i--)
//		{
//			bool swap = false;
//			for (int j = 0; j < i; j++)
//			{
//				if (dist[j] > dist[j + 1])
//				{
//					float temp_dist = dist[j];
//					dist[j] = dist[j + 1];
//					dist[j + 1] = temp_dist;
//					swap = true;
//				}
//			}
//			if (!swap)
//			{
//				break;
//			}
//		}
//
//		int i = 0, j = 0;
//		while ((i < 10) && (j < 10))
//		{
//			if ((last[2 * i] == FLT_MAX) || (last[2 * i + 1] == FLT_MAX) || (dist[2 * j] == FLT_MAX) || (dist[2 * j + 1] == FLT_MAX))
//			{
//				break;
//			}
//			if ((last[2 * i + 1] > dist[2 * j]) && (last[2 * i] < dist[2 * j + 1]))
//			{
//				if (map_new[j] == 10)
//				{
//					map_new[j] = map[i];
//					areas[map_new[j]] += dist[2 * j + 1] - dist[2 * j];
//				}
//				else if (map_new[j] != map[i])
//				{
//					areas[map_new[j]] += areas[map[i]];
//				}
//
//			}
//			if (last[2 * i + 1] < dist[2 * j + 1])
//			{
//				i++;
//			}
//			else if (last[2 * i + 1] > dist[2 * j + 1])
//			{
//				j++;
//			}
//			else
//			{
//				i++;
//				j++;
//			}
//		}
//		for (int i = 0; i < 10; i++)
//		{
//			last[2 * i] = dist[2 * i];
//			last[2 * i + 1] = dist[2 * i + 1];
//			if (map_new[i] != 10)
//			{
//				map[i] = map_new[i];
//			}
//		}
//	}
//	float output = 0;
//	for (int i = 0; i < 10; i++)
//	{
//		output = (areas[i] > output) ? areas[i] : output;
//	}
//
//	return output;
//}

__host__ __device__ void poly::point_get(point*& ��) 
{
	changed();

	if (�� != nullptr)
	{
		delete[]��;
	}
	�� = new point[20];
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

void poly::point_get(std::vector<point>& ��) 
{
	changed();

	�� = std::vector<point>(20);
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

__host__ __device__ void poly::seg_get(seg*& �߶�) 
{
	changed();

	if (�߶� != nullptr)
	{
		delete[]�߶�;
	}
	�߶� = new seg[20];
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

void poly::seg_get(std::vector<seg>& �߶�) 
{
	changed();

	�߶� = std::vector<seg>(20);
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

__host__ __device__ bool poly::point_in(point ��) const
{
	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20 ; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
	}
	if ((max[0] < ��[0]) || (max[1] < ��[1]) || (min[0] > ��[0]) || (min[1] > ��[1]))
	{
		return false;
	}

	ray temp;
	temp.origin = ��;
	temp.dir = vector(0.0f, 1.0f);
	int k = 0;

	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}

	temp.dir = vector(point({ 0,-1 }));
	k = 0;
	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}
	return true;
}

__host__ __device__ void poly::reset_seg()
{
	if (!seg_change)
	{
		return;
	}
	seg_change = false;

	area_change = true;
	center_change = true;
	legal_change = true;

	for (int i = 0, n = 0; (i < 20 - 1) && (n < 20); i++)
	{
		if ((abs(segs[i].origin[0] - segs[i + 1].origin[0]) > 0.001f) || (abs(segs[i].origin[1] - segs[i + 1].origin[1]) > 0.001f))
		{
			continue;
		}
		n++;
		i--;
		for (int j = i + 1; j < 20 - 1; j++)
		{
			segs[j].origin = segs[j + 1].origin;
		}
		segs[19].origin = segs[0].origin;
	}
	


	for (int i = 0; i < 20 - 1; i++)
	{
		segs[i] = seg(segs[i].origin, segs[i + 1].origin);
	}
	segs[19] = seg(segs[19].origin, segs[0].origin);
}

__host__ __device__ void poly::reset_seg(int i)
{
	if (!seg_change)
	{
		return;
	}

	area_change = true;
	center_change = true;
	legal_change = true;

	segs[i] = seg(segs[i].origin, segs[(i + 1) % 20].origin);
}

__host__ __device__ seg& poly::operator[](int i)
{
	changed();

	while (i < 0)
	{
		i += 20;
	}
	return segs[i % 20];
}

__host__ __device__ seg poly::operator[](int i) const
{
	while (i < 0)
	{
		i += 20;
	}
	return segs[i % 20];
}

__host__ __device__ float poly::dir_area() const
{
	if (!dir_area_change)
	{
		return dir_area_;
	}
	dir_area_change = false;
	float s = 0;
	for (int i = 0; i < 20 - 1; i++)
	{
		s += vector(segs[i].origin) ^ vector(segs[i + 1].origin);
	}
	s += vector(segs[20 - 1].origin) ^ vector(segs[0].origin);


	dir_area_ = s / 2;
	return s / 2;
}


__global__ void poly_area(seg* segs, float min_x, float min_y, float max_x, float max_y, float* output)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int x = min_x + idx;
	if (x >= max_x)
	{
		return;
	}
	float min[2] = { min_x,min_y };
	float max[2] = { max_x,max_y };

	seg temp;
	temp.origin = point(x, min[1]);
	temp.dir = vector(0.0f, 1.0f);
	temp.dist = max[1] - min[1];

	float dist[20];

	for (int i = 0; i < 20; i++)
	{
		float t_1, t_2;
		cross(temp, segs[i], t_1, t_2);
		dist[i] = t_1;
	}
	sort(dist, 20);
	//for (int i = 19; i > 0; i--)
	//{
	//	bool swap = false;
	//	for (int j = 0; j < i; j++)
	//	{
	//		if (dist[j] > dist[j + 1])
	//		{
	//			float temp_dist = dist[j];
	//			dist[j] = dist[j + 1];
	//			dist[j + 1] = temp_dist;
	//			swap = true;
	//		}
	//	}
	//	if (!swap)
	//	{
	//		break;
	//	}
	//}

	output[idx] = 0;
	for (int i = 0; i < 10; i++)
	{
		if ((dist[2 * i + 1] == FLT_MAX) || (dist[2 * i] == FLT_MAX))
		{
			break;
		}
		output[idx] += dist[2 * i + 1] - dist[2 * i];
	}
}

__host__ __device__ float poly::area() const
{
	if (!area_change)
	{
		return area_;
	}
	area_change = false;
	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
	}

#ifndef __HIPCC__
	int device_n;
	hipGetDeviceCount(&device_n);

	if (((max[0] - min[0]) > 100) && (device_n > 0))
	{
		seg* segs_d = NULL;
		hipMalloc((void**)&segs_d, sizeof(seg) * 20);
		hipMemcpy(segs_d, segs, sizeof(seg) * 20, hipMemcpyHostToDevice);
		float* output_d = NULL;
		hipMalloc((void**)&output_d, sizeof(float) * int(max[0] - min[0]));

		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		int ÿ���߳� = deviceProp.maxThreadsPerBlock / 32;
		int �� = int(max[0] - min[0]) / ÿ���߳� + 1;

		poly_area << < ��, ÿ���߳� >> > (segs_d, min[0], min[1], max[0], max[1], output_d);
		hipFree(segs_d);

		float* output_h = new float[int(max[0] - min[0])];
		hipMemcpy(output_h, output_d, sizeof(float) * int(max[0] - min[0]), hipMemcpyDeviceToHost);
		hipFree(output_d);
		float output = 0;
		for (int i = 0; i<int(max[0] - min[0]); i++)
		{
			output += output_h[i];
		}
		delete[]output_h;
		area_ = output;
		return output;
	}

#endif


	float output = 0;
	for (int x = min[0]; x < max[0]; x++)
	{
		seg temp;
		temp.origin = point(x, min[1]);
		temp.dir = vector(0.0f, 1.0f);
		temp.dist = max[1] - min[1];

		float dist[20];

		for (int i = 0; i < 20; i++)
		{
			float t_1, t_2;
			cross(temp, segs[i], t_1, t_2);
			dist[i] = t_1;
		}
		sort(dist, 20);
		//for (int i = 19; i > 0; i--)
		//{
		//	bool swap = false;
		//	for (int j = 0; j < i; j++)
		//	{
		//		if (dist[j] > dist[j + 1])
		//		{
		//			float temp_dist = dist[j];
		//			dist[j] = dist[j + 1];
		//			dist[j + 1] = temp_dist;
		//			swap = true;
		//		}
		//	}
		//	if (!swap)
		//	{
		//		break;
		//	}
		//}

		for (int i = 0; i < 10; i++)
		{
			if ((dist[2 * i + 1] == FLT_MAX) || (dist[2 * i] == FLT_MAX))
			{
				break;
			}
			output += dist[2 * i + 1] - dist[2 * i];
		}
	}
	area_ = output;
	return output;
}
#ifndef no_opencv
void poly::print(cv::InputOutputArray ͼ��, float ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	//seg(segs[0].origin, segs[1].origin).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	for (int i = 0; i < 19; i++)
	{
		seg(segs[i].origin, segs[i + 1].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	}
	seg(segs[19].origin, segs[0].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	//segs[0].origin.print(ͼ��, ����, ��ɫ, ��ϸ * 4);
}
#endif
__global__ void poly_center(seg* segs, float min_x, float min_y, float max_x, float max_y, float* p_area, float* x_, float* y_)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int x = min_x + idx;
	if (x >= max_x)
	{
		return;
	}
	float min[2] = { min_x,min_y };
	float max[2] = { max_x,max_y };

	seg temp;
	temp.origin = point(x, min[1]);
	temp.dir = vector(0.0f, 1.0f);
	temp.dist = max[1] - min[1];

	float dist[20];

	for (int i = 0; i < 20; i++)
	{
		float t_1, t_2;
		cross(temp, segs[i], t_1, t_2);
		dist[i] = t_1;
	}
	sort(dist, 20);
	//for (int i = 19; i > 0; i--)
	//{
	//	bool swap = false;
	//	for (int j = 0; j < i; j++)
	//	{
	//		if (dist[j] > dist[j + 1])
	//		{
	//			float temp_dist = dist[j];
	//			dist[j] = dist[j + 1];
	//			dist[j + 1] = temp_dist;
	//			swap = true;
	//		}
	//	}
	//	if (!swap)
	//	{
	//		break;
	//	}
	//}

	p_area[idx] = 0, x_[idx] = 0, y_[idx] = 0;
	for (int i = 0; i < 10; i++)
	{
		if ((dist[2 * i + 1] == FLT_MAX) || (dist[2 * i] == FLT_MAX))
		{
			break;
		}
		p_area[idx] += dist[2 * i + 1] - dist[2 * i];
		y_[idx] += powf(dist[2 * i + 1] + min[1], 2) - powf(dist[2 * i] + min[1], 2);
	}
	x_[idx] = p_area[idx] * x;
}

__host__ __device__ point poly::center() const
{
	if (!center_change)
	{
		return center_;
	}
	center_change = false;

	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
	}


#ifndef __HIPCC__
	int device_n;
	hipGetDeviceCount(&device_n);

	if (((max[0] - min[0]) > 100) && (device_n > 0))
	{
		seg* segs_d = NULL;//
		hipMalloc((void**)&segs_d, sizeof(seg) * 20);
		hipMemcpy(segs_d, segs, sizeof(seg) * 20, hipMemcpyHostToDevice);
		float* p_area_d = NULL;//
		hipMalloc((void**)&p_area_d, sizeof(float) * int(max[0] - min[0]));
		float* x_d = NULL;//
		hipMalloc((void**)&x_d, sizeof(float) * int(max[0] - min[0]));
		float* y_d = NULL;//
		hipMalloc((void**)&y_d, sizeof(float) * int(max[0] - min[0]));

		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		int ÿ���߳� = deviceProp.maxThreadsPerBlock / 32;
		int �� = int(max[0] - min[0]) / ÿ���߳� + 1;

		poly_center << < ��, ÿ���߳� >> > (segs_d, min[0], min[1], max[0], max[1], p_area_d, x_d, y_d);
		hipFree(segs_d);

		float* p_area_h = new float[int(max[0] - min[0])];
		hipMemcpy(p_area_h, p_area_d, sizeof(float) * int(max[0] - min[0]), hipMemcpyDeviceToHost);
		float* x_h = new float[int(max[0] - min[0])];
		hipMemcpy(x_h, x_d, sizeof(float) * int(max[0] - min[0]), hipMemcpyDeviceToHost);
		float* y_h = new float[int(max[0] - min[0])];
		hipMemcpy(y_h, y_d, sizeof(float) * int(max[0] - min[0]), hipMemcpyDeviceToHost);
		hipFree(p_area_d);
		hipFree(x_d);
		hipFree(y_d);
		float p_area = 0, x = 0, y = 0;
		for (int i = 0; i<int(max[0] - min[0]); i++)
		{
			p_area += p_area_h[i];
			x += x_h[i];
			y += y_h[i];
		}
		delete[]p_area_h;
		delete[]x_h;
		delete[]y_h;
		center_= point(x / p_area, y / 2 / p_area);
		return point(x / p_area, y / 2 / p_area);
	}

#endif


	float p_area = 0, x_ = 0, y_ = 0;
	for (int x = min[0]; x < max[0]; x++)
	{
		seg temp;
		temp.origin = point(x, min[1]);
		temp.dir = vector(0.0f, 1.0f);
		temp.dist = max[1] - min[1];

		float dist[20];

		for (int i = 0; i < 20; i++)
		{
			float t_1, t_2;
			cross(temp, segs[i], t_1, t_2);
			dist[i] = t_1;
		}
		sort(dist, 20);

		float d_x = 0;
		for (int i = 0; i < 10; i++)
		{
			if ((dist[2 * i + 1] == FLT_MAX) || (dist[2 * i] == FLT_MAX))
			{
				break;
			}
			p_area += dist[2 * i + 1] - dist[2 * i];
			d_x += dist[2 * i + 1] - dist[2 * i];
			y_ += powf(dist[2 * i + 1] + min[1], 2) - powf(dist[2 * i] + min[1], 2);
		}
		x_ += d_x * x;
	}

	center_ = point(x_ / p_area, y_ / 2 / p_area);
	return point(x_ / p_area, y_ / 2 / p_area);
}

__host__ __device__ point poly::fast_center() const
{
	if (!fast_center_change)
	{
		return fast_center_;
	}
	fast_center_change = false;

	float s = 0;
	point center__(0.0f, 0.0f);
	for (int i = 0; i < 19; i++)
	{
		float a = vector(segs[i].origin) ^ vector(segs[i + 1].origin);
		s += a;
		center__ = point(vector(center__) + a * (vector(segs[i].origin) + vector(segs[i + 1].origin)));
	}
	{
		float a = vector(segs[19].origin) ^ vector(segs[0].origin);
		s += a;
		center__ = point(vector(center__) + a * (vector(segs[19].origin) + vector(segs[0].origin)));
	}
	center__ = point(vector(center__) / s / 3);
	fast_center_ = center__;
	return center__;
}

vector poly::move2center()
{
	vector move(fast_center());

	for (int i = 0; i < 20; i++)
	{
		segs[i].origin = point(vector(segs[i].origin) - move);
	}

	center_ = point(0.0, 0.0);
	fast_center_ = point(0.0, 0.0);

	return vector(0.0f, 0.0f) - move;
}

__host__ __device__ void poly::simple(float �Ƕ�, bool rad)
{
	changed();

	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	float cos_ = cos(�Ƕ�);

	reset_seg();
	int n = 1;
	while (n != 0)	{

		n = 0;
		for (int i = 0, j = 1; j < 20; j++)
		{
			i = j - 1;

			float cos_t = (vector(0.0f, 0.0f) - segs[i].dir) * segs[j].dir;
			if ((cos_t < cos_) || (segs[i].dist < 0.0001f) || (segs[j].dist < 0.0001f))
			{
				continue;
			}

			n++;
			if (i == 18)
			{
				segs[19].origin = segs[0].origin;
				changed();
			}

			for (int k = i + 1; k < 20 - 1; k++)
			{
				segs[k].origin = segs[k + 1].origin;
				changed();
			}
			reset_seg();
		}

		vector dir_;
		for (int i = 19; i >= 0; i--)
		{
			if (segs[i].dist > 0.0001f)
			{
				dir_ = segs[i].dir;
				break;
			}
		}

		float cos_t = (vector(0.0f, 0.0f) - dir_) * segs[0].dir;
		if (cos_t < cos_)
		{
			continue;
		}

		n++;
		for (int j = 0; j < 20 - 1; j++)
		{
			segs[j].origin = segs[j + 1].origin;
			changed();
		}
		reset_seg();
	}
}

__host__ __device__ bool poly::is_overlap(const poly other) const
{
	return ::is_overlap(*this, other);
}

__host__ __device__ bool poly::full_overlap(const poly other) const
{
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			if (is_cross(other[i], segs[j]))
			{
				return false;
			}
		}
		if (!point_in(other[i].origin))
		{
			return false;
		}
	}
	return true;
}

__host__ __device__ float poly::overlap_area(const poly other) const
{
	return ::overlap_area(*this, other);
}

__host__ __device__ bool is_overlap(const poly p_1, const poly p_2)
{
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			if (is_cross(p_1[i], p_2[j]))
			{
				return true;
			}
		}
		if (p_1.point_in(p_2[i].origin))
		{
			return true;
		}
		if (p_2.point_in(p_1[i].origin))
		{
			return true;
		}
	}
	return false;
}


__global__ void overlap_area_cuda(poly* p, float min_x, float min_y, float max_x, float max_y, float* output)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int i = min_x + idx;
	if (i >= max_x)
	{
		return;
	}
	float min[2] = { min_x,min_y };
	float max[2] = { max_x,max_y };
	poly p_1 = p[0], p_2 = p[1];

	ray temp;
	temp.origin = point(i, min[1]);
	temp.dir = vector(0.0f, 1.0f);


	bool in_1 = false, in_2 = false;
	float dist[2][20];
	for (int j = 0; j < 20; j++)
	{
		float t_1, t_2;
		cross(temp, p_1[j], t_1, t_2);
		dist[0][j] = t_1;
		if (t_1 != FLT_MAX)
		{
			in_1 = !in_1;
		}
		cross(temp, p_2[j], t_1, t_2);
		dist[1][j] = t_1;
		if (t_1 != FLT_MAX)
		{
			in_2 = !in_2;
		}
	}
	sort(dist[0], 20);
	sort(dist[1], 20);

	int j = 0, k = 0;
	while ((j < 20) && (k < 20))
	{
		float next_1 = min[1] + dist[0][j] - temp.origin[1], next_2 = min[1] + dist[1][k] - temp.origin[1];

		if (in_1 && in_2 && ((next_1 < (max[1] - min[1])) || (next_2 < (max[1] - min[1]))))
		{
			output[idx] += fmin(next_1, next_2);
		}
		else if ((next_1 > (max[1] - min[1])) || (next_2 > (max[1] - min[1])))
		{
			break;
		}
		if (next_1 < next_2)
		{
			j++;
			in_1 = !in_1;
		}
		else if (next_1 > next_2)
		{
			k++;
			in_2 = !in_2;
		}
		else if(next_1 == next_2)
		{
			j++;
			k++;
			in_1 = !in_1;
			in_2 = !in_2;
		}

	}
}

__host__ __device__ float overlap_area(const poly p_1, const poly p_2)
{
	point max_1 = p_1[0].origin, min_1 = p_1[0].origin;
	for (int i = 1; i < 20; i++)
	{
		max_1[0] = (max_1[0] > p_1[i].origin[0]) ? max_1[0] : p_1[i].origin[0];
		max_1[1] = (max_1[1] > p_1[i].origin[1]) ? max_1[1] : p_1[i].origin[1];
		min_1[0] = (min_1[0] < p_1[i].origin[0]) ? min_1[0] : p_1[i].origin[0];
		min_1[1] = (min_1[1] < p_1[i].origin[1]) ? min_1[1] : p_1[i].origin[1];
	}
	point max_2 = p_2[0].origin, min_2 = p_2[0].origin;
	for (int i = 0; i < 20; i++)
	{
		max_2[0] = (max_2[0] > p_2[i].origin[0]) ? max_2[0] : p_2[i].origin[0];
		max_2[1] = (max_2[1] > p_2[i].origin[1]) ? max_2[1] : p_2[i].origin[1];
		min_2[0] = (min_2[0] < p_2[i].origin[0]) ? min_2[0] : p_2[i].origin[0];
		min_2[1] = (min_2[1] < p_2[i].origin[1]) ? min_2[1] : p_2[i].origin[1];
	}

	point max(fmin(max_1[0], max_2[0]), fmin(max_1[1], max_2[1])), min(fmax(min_1[0], min_2[0]), fmax(min_1[1], min_2[1]));


#ifndef __HIPCC__
	int device_n;
	hipGetDeviceCount(&device_n);

	if (((max[0] - min[0]) > 100) && (device_n > 0))
	{
		poly p_h[2] = { p_1,p_2 };
		poly* p_d = NULL;
		hipMalloc((void**)&p_d, sizeof(poly) * 2);
		hipMemcpy(p_d, p_h, sizeof(poly) * 2, hipMemcpyHostToDevice);

		float* output_d = NULL;
		hipMalloc((void**)&output_d, sizeof(float) * int(max[0] - min[0]));

		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		int ÿ���߳� = deviceProp.maxThreadsPerBlock / 32;
		int �� = int(max[0] - min[0]) / ÿ���߳� + 1;

		overlap_area_cuda << < ��, ÿ���߳� >> > (p_d, min[0], min[1], max[0], max[1], output_d);
		hipFree(p_d);

		float* output_h = new float[int(max[0] - min[0])];
		hipMemcpy(output_h, output_d, sizeof(float) * int(max[0] - min[0]), hipMemcpyDeviceToHost);
		hipFree(output_d);
		float output = 0;
		for (int i = 0; i<int(max[0] - min[0]); i++)
		{
			output += output_h[i];
		}
		delete[]output_h;
		return output;
	}

#endif

	float output = 0;
	for (int i = min[0]; i < max[0]; i++)
	{
		ray temp;
		temp.origin = point(i, min[1]);
		temp.dir = vector(0.0f, 1.0f);

		
		bool in_1 = false, in_2 = false;
		float dist[2][20];
		for (int j = 0; j < 20; j++)
		{
			float t_1, t_2;
			cross(temp, p_1[j], t_1, t_2);
			dist[0][j] = t_1;
			if (t_1 != FLT_MAX)
			{
				in_1 = !in_1;
			}
			cross(temp, p_2[j], t_1, t_2);
			dist[1][j] = t_1;
			if (t_1 != FLT_MAX)
			{
				in_2 = !in_2;
			}
		}
		sort(dist[0], 20);
		sort(dist[1], 20);

		int j = 0, k = 0;
		while ((j < 20) && (k < 20))
		{
			float next_1 = min[1] + dist[0][j] - temp.origin[1], next_2 = min[1] + dist[1][k] - temp.origin[1];

			if (in_1 && in_2 && ((next_1 < (max[1] - min[1])) || (next_2 < (max[1] - min[1]))))
			{
				output += fmin(next_1, next_2);
			}
			else if ((next_1 > (max[1] - min[1])) || (next_2 > (max[1] - min[1])))
			{
				break;
			}
			if (next_1 < next_2)
			{
				j++;
				in_1 = !in_1;
			}
			else if (next_1 > next_2)
			{
				k++;
				in_2 = !in_2;
			}
			else if (next_1 == next_2)
			{
				j++;
				k++;
				in_1 = !in_1;
				in_2 = !in_2;
			}
		}
	}
	return output;
}

__host__ __device__ float dist(const poly p_1, const poly p_2)
{
	return length(p_1.fast_center(), p_2.fast_center());
}

__host__ __device__ float dist(const poly p, const line l)
{
	float d = FLT_MAX;
	for (int i = 0; i < 20; i++)
	{
		if (is_cross(p[i], l))
		{
			return 0;
		}
		float t = l.point_dist(p[i].origin);
		if (t < d)
		{
			d = t;
		}
	}
	return d;
}

__host__ __device__ float dist(const poly p, const ray l)
{
	float d = FLT_MAX;
	for (int i = 0; i < 20; i++)
	{
		if (is_cross(p[i], l))
		{
			return 0;
		}
		float t = l.point_dist(p[i].origin);
		if (t < d)
		{
			d = t;
		}
	}
	return d;
}

__host__ __device__ float dist(const poly p, const seg l)
{
	float d = FLT_MAX;
	for (int i = 0; i < 20; i++)
	{
		if (is_cross(p[i], l))
		{
			return 0;
		}
		float t = l.point_dist(p[i].origin);
		if (t < d)
		{
			d = t;
		}
	}
	return d;
}
