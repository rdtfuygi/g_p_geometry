#include "hip/hip_runtime.h"
#include "geometry.cuh"
#define _USE_MATH_DEFINES 
#include <math.h>

__host__ __device__ vector::vector() :point(1, 0) {}

__host__ __device__ vector::vector(float x, float y) : point(x, y) {}

__host__ __device__ vector::vector(point ��) : point(��) {};

__host__ __device__ vector::vector(float ����[2], float ����)
{
	float ���� = ���� / ::length({ 0,0 }, ����);
	locat[0] = ����[0] * ����;
	locat[1] = ����[1] * ����;
}

__host__ __device__ vector::vector(float �Ƕ�, bool rad, float ����)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	locat[0] = cos(�Ƕ�) * ����;
	locat[1] = sin(�Ƕ�) * ����;
}

__host__ __device__ vector& vector::operator+=(vector ����)
{
	locat[0] += ����[0];
	locat[1] += ����[1];
	return *this;
}

__host__ __device__ vector& vector::operator-=(vector ����)
{
	locat[0] -= ����[0];
	locat[1] -= ����[1];
	return *this;
}

__host__ __device__ vector& vector::operator*=(float ��)
{
	locat[0] *= ��;
	locat[1] *= ��;
	return *this;
}

__host__ __device__ vector& vector::operator/=(float ��)
{
	locat[0] /= ��;
	locat[1] /= ��;
	return *this;
}

__host__ __device__ vector vector::unitize() const
{
	float ���� = length();
	if (���� < 1e-16)
	{
		return vector(float(M_SQRT1_2), float(M_SQRT1_2));
	}
	return vector(*this / ����);
}

__host__ __device__ float vector::length() const
{
	return ::length(*this);
}

__host__ __device__ vector vector::rotate(float �Ƕ�, bool rad) const
{
	return vector(::rotate({ 0,0 }, point(*this), �Ƕ�, rad));
}

__host__ __device__ float vector::angle_get(bool rad) const
{
	float �Ƕ� = atan(locat[1] / locat[0]) + (locat[0] > 0 ? 0 : M_PI);
	if (!rad)
	{
		�Ƕ� = rad2deg(�Ƕ�);
	}
	return �Ƕ�;
}

void vector::print(cv::InputOutputArray ͼ��, float ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	int �� = ͼ��.rows(), �� = ͼ��.cols();
	int ԭ��_x = �� / 2, ԭ��_y = �� / 2;

	//int �Ŵ� = 2 * (�� > �� ? �� : ��);

	cv::Point ��_1(ԭ��_x, ԭ��_y);
	cv::Point ��_2(locat[0] * ���� + ԭ��_x, -locat[1] * ����  + ԭ��_y);
	cv::line(ͼ��, ��_1, ��_2, ��ɫ, ��ϸ);
}

__host__ __device__ float inc_angle_cos(vector ����_1, vector ����_2)
{	
	return ����_1.unitize() * ����_2.unitize();
}

__host__ __device__ float inc_angle_sin(vector ����_1, vector ����_2)
{
	return ����_1.unitize() ^ ����_2.unitize();
}

__host__ __device__ vector operator-(vector ����)
{
	return vector(-����[0], -����[1]);
}

__host__ __device__ vector operator+(vector ����_1, vector ����_2)
{
	return vector(����_1[0] + ����_2[0], ����_1[1] + ����_2[1]);
}

__host__ __device__ vector operator-(vector ����_1, vector ����_2)
{
	return vector(����_1[0] - ����_2[0], ����_1[1] - ����_2[1]);
}

__host__ __device__ vector operator*(vector ����, float ��)
{
	return vector(����[0] * ��, ����[1] * ��);
}

__host__ __device__ vector operator*(float ��, vector ����)
{
	return vector(����[0] * ��, ����[1] * ��);
}

__host__ __device__ vector operator/(vector ����, float ��)
{
	return vector(����[0] / ��, ����[1] / ��);
}


__host__ __device__ float length(vector ����)
{
	return length({ 0,0 }, ����);
}



__host__ __device__ float operator*(vector ����_1, vector ����_2)
{
	return ����_1[0] * ����_2[0] + ����_1[1] * ����_2[1];
}

__host__ __device__ float operator^(vector ����_1, vector ����_2)
{
	return (����_1[0] * ����_2[1]) - (����_1[1] * ����_2[0]);
}